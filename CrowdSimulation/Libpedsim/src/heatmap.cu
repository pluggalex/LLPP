#include "hip/hip_runtime.h"
// Created for Low Level Parallel Programming 2017
//
// Implements the heatmap functionality. 
//
//#include "heatmap.h"
#include "ped_model.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <math.h>


// Sets up the heatmap
void Ped::Model::setupHeatmapSeq()
{
	int *hm = (int*)calloc(SIZE*SIZE, sizeof(int));
	int *shm = (int*)malloc(SCALED_SIZE*SCALED_SIZE*sizeof(int));
	int *bhm = (int*)malloc(SCALED_SIZE*SCALED_SIZE*sizeof(int));

	heatmap = (int**)malloc(SIZE*sizeof(int*));

	scaled_heatmap = (int**)malloc(SCALED_SIZE*sizeof(int*));
	blurred_heatmap = (int**)malloc(SCALED_SIZE*sizeof(int*));

	for (int i = 0; i < SIZE; i++)
	{
		heatmap[i] = hm + SIZE*i;
	}
	for (int i = 0; i < SCALED_SIZE; i++)
	{
		scaled_heatmap[i] = shm + SCALED_SIZE*i;
		blurred_heatmap[i] = bhm + SCALED_SIZE*i;
	}

	//CUDA init
	//enum hipError_t malloc_status = hipMallocPitch((void**)&d_heatmap, pitch, SIZE*sizeof(int), SIZE*sizeof(int));
	//enum hipError_t memcpy_status = hipMemcpy(d_heatmap, heatmap, SIZE*SIZE*sizeof(int), hipMemcpyHostToDevice);

	enum hipError_t malloc_status = hipMalloc((void**)&d_heatmap, SIZE*SIZE*sizeof(int));
	malloc_status = hipMalloc((void**)&d_heatmap_row_size, sizeof(int));
	enum hipError_t memset_status = hipMemset(d_heatmap_row_size, SIZE, sizeof(int));

	malloc_status = hipMalloc((void**)&d_scaled_heatmap, SIZE*SIZE*CELLSIZE*CELLSIZE*sizeof(int));
	memset_status = hipMemset(d_scaled_heatmap, 0, SIZE*SIZE*CELLSIZE*CELLSIZE*sizeof(int));

	malloc_status = hipMalloc((void**)&d_scaled_heatmap_row_size, sizeof(int));
	memset_status = hipMemset(d_scaled_heatmap_row_size, SIZE*CELLSIZE, sizeof(int));
	
}

/*
for (int x = 0; x < SIZE; x++)
{
for (int y = 0; y < SIZE; y++)
{
// heat fades
heatmap[y][x] = (int)round(heatmap[y][x] * 0.80);
}
}
*/
__global__
void fade(int* d_heatmap){
	// heat fades
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	d_heatmap[tid] = __double2int_rd(d_heatmap[tid] * 0.80);
}

__global__
void locationContention(int* d_heatmap, float* d_desiredXs, float* d_desiredYs, int* d_agents){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < *d_agents){
		int desX = d_desiredXs[tid];
		int desY = d_desiredYs[tid];
		int inc = 40;
		atomicAdd(&d_heatmap[desY*1024 + desX], inc);
		//d_heatmap[y][x] += 40 ;
	}
}

__global__
void ceiling(int* d_heatmap){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int ceiling = 255;
	if (d_heatmap[tid] > ceiling)
		d_heatmap[tid] = ceiling;
}

__global__
void scale(int* d_scaled_heatmap, int* d_scaled_heatmap_row_size, int* d_heatmap, int* d_heatmap_row_size){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int cellsize = 5;
	int s_row_size = 1024 * 5;
	int s_row = __double2int_rd(tid / s_row_size);//__double2int_rd(blockIdx.x / cellsize);
	int s_col = tid;
	if (s_row)
		s_col = tid - s_row * s_row_size;

	int row = s_row / cellsize;
	int col = s_col / cellsize;
	int index = col + row * 1024;// (*d_heatmap_row_size);
	d_scaled_heatmap[tid] = d_heatmap[index];

	//if (tid == 1024)
	//	printf("s_row: %d, s_col: %d, row: %d, col:%d, s_row_size:%d\n", s_row, s_col, row, col, s_row_size);
}

// Updates the heatmap according to the agent positions
void Ped::Model::updateHeatmapSeq()
{
	//INIT d_heatmap
	enum hipError_t memcpy_status = hipMemcpy(d_heatmap, heatmap[0], SIZE*SIZE*sizeof(int), hipMemcpyHostToDevice);
	int threadsPerBlock = 512;
	int blocks = SIZE*SIZE / threadsPerBlock;
	
	//Fade heatmap
	fade << <blocks, threadsPerBlock >> >(d_heatmap);
	enum hipError_t sync_status = hipDeviceSynchronize();
	
	//Init desireds for cuda
	vector<float> tempx = agentCollection->getDesiredX();
	vector<float> tempy = agentCollection->getDesiredY();
	float* Xs = &(tempx[0]);
	float* Ys = &(tempy[0]);
	int no_agents = agentCollection->size();
	
	int* d_size;
	hipMalloc(&d_size, sizeof(int));
	hipMemcpy(d_size, &no_agents, sizeof(int), hipMemcpyHostToDevice);

	float* d_desiredXs;// = &(Xs[0]);
	hipMalloc(&d_desiredXs, no_agents*sizeof(float));
	hipMemcpy(d_desiredXs, Xs, no_agents*sizeof(float), hipMemcpyHostToDevice);
	
	float* d_desiredYs;// = &(Ys[0]);	
	hipMalloc(&d_desiredYs, no_agents*sizeof(float));
	hipMemcpy(d_desiredYs, Ys, no_agents*sizeof(float), hipMemcpyHostToDevice);

	//Set location contention based on desired
	locationContention <<<blocks, threadsPerBlock>>>(d_heatmap, d_desiredXs, d_desiredYs, d_size);
	sync_status = hipDeviceSynchronize();
	
	//Cut of values to max 255
	ceiling<< <blocks, threadsPerBlock >> >(d_heatmap);
	hipMemcpy(heatmap[0], d_heatmap, SIZE*SIZE*sizeof(int), hipMemcpyDeviceToHost);
	sync_status = hipDeviceSynchronize();

	// Scale the data for visual representation
	blocks = SIZE*SIZE*CELLSIZE*CELLSIZE / threadsPerBlock;
	scale << <blocks, threadsPerBlock >> >(d_scaled_heatmap, d_scaled_heatmap_row_size, d_heatmap, d_heatmap_row_size);
	hipMemcpy(scaled_heatmap[0], d_scaled_heatmap, SIZE*SIZE*CELLSIZE*CELLSIZE*sizeof(int), hipMemcpyDeviceToHost);
	sync_status = hipDeviceSynchronize();
	
	// Weights for blur filter
	const int w[5][5] = {
		{ 1, 4, 7, 4, 1 },
		{ 4, 16, 26, 16, 4 },
		{ 7, 26, 41, 26, 7 },
		{ 4, 16, 26, 16, 4 },
		{ 1, 4, 7, 4, 1 }
	};
	int val = 0;
#define WEIGHTSUM 273
	// Apply gaussian blurfilter		       
	for (int i = 2; i < SCALED_SIZE - 2; i++)
	{
		for (int j = 2; j < SCALED_SIZE - 2; j++)
		{
			int sum = 0;
			for (int k = -2; k < 3; k++)
			{
				for (int l = -2; l < 3; l++)
				{
					//int val =  scaled_heatmap[i + k][j + l];
					//if (val != 0)
					//	printf("123");
					sum += w[2 + k][2 + l] * scaled_heatmap[i + k][j + l];
				}
			}
			int value = sum / WEIGHTSUM;
			blurred_heatmap[i][j] = 0x00FF0000 | value << 24;
		}
	}
}



int Ped::Model::getHeatmapSize() const {
	return SCALED_SIZE;
}

